#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Copyright (c) 2017
// The Chinese University of Hong Kong
// Written by Hu Xiaowei
//
// ShiftLeftLayer shift the feature map 
// ------------------------------------------------------------------
#include <vector>

#include "caffe/layers/shift_left_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>  //left up right down
__global__ void ShiftLeftForward(const int nthreads, const int stride, const Dtype* bottom_data, Dtype* top_left, const int channel, const int height, const int width) {
  CUDA_KERNEL_LOOP(index, nthreads) {

     int w = index % width;
     int h = (index / width) % height;
     int c = (index / width / height) % channel;
     int n = index / width / height / channel;

     if ((w+stride)<width)
     {
        top_left[index] = bottom_data[((n*channel + c)*height + h)*width + w + stride];
     }
     else
     {
        top_left[index] = 0; //bottom_data[((n*channel + c)*height + h)*width + w];
     }
     
  }
}

template <typename Dtype>
__global__ void ShiftLeftBackward(const int nthreads, const int stride, Dtype* bottom_diff, const Dtype* top_left, const int channel, const int height, const int width) {
  CUDA_KERNEL_LOOP(index, nthreads) {
     
     int w = index % width;
     int h = (index / width) % height;
     int c = (index / width / height) % channel;
     int n = index / width / height / channel;

     Dtype diff = 0;
 

     if ((w-stride)>=0)
     {
        diff += top_left[((n*channel + c)*height + h)*width + w - stride];
     }
     else
     {
        diff += 0; //top_left[((n*channel + c)*height + h)*width + w];
     }

     bottom_diff[index] = diff;

  }
}

template <typename Dtype>
void ShiftLeftLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  
  int count = bottom[0]->count();
 
  Dtype* top_data = top[0]->mutable_gpu_data();

  const Dtype* bottom_data = bottom[0]->gpu_data();

  ShiftLeftForward<Dtype>
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, stride, bottom_data, top_data, channels_, height_, width_);
  CUDA_POST_KERNEL_CHECK;

}

template <typename Dtype>
void ShiftLeftLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {

  int count = bottom[0]->count();

  const Dtype* top_left = top[0]->gpu_diff();

  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();

  ShiftLeftBackward<Dtype>  // NOLINT_NEXT_LINE(whitespace/operators)
        <<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(count, stride, bottom_diff, top_left, channels_, height_, width_);
  CUDA_POST_KERNEL_CHECK;

}

INSTANTIATE_LAYER_GPU_FUNCS(ShiftLeftLayer);

}  // namespace caffe
